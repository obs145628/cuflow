#include "hip/hip_runtime.h"
#include "matmul.hh"
#include "../app/mode.hh"
#include "../app/timer.hh"
#include <cmath>
#include <math_functions.h>

namespace gpu
{

    namespace
    {

        constexpr std::size_t BLOCK_SIZE = 512;

        __global__
        void log_softmax1(const float* x, float* y,
                      std::size_t rows, std::size_t cols) //8ms
        {
            __shared__ float partial[2 * BLOCK_SIZE];

            //load all elements of the array in shared memory
            auto row = blockIdx.x;
            auto col = threadIdx.x;
            std::size_t step = BLOCK_SIZE;

            float init = 1e-30;
            for (std::size_t i = col; i < cols; i += step)
                init = max(x[row * cols + i], init);
        
            partial[col] = init;
            __syncthreads();

            for (std::size_t s = BLOCK_SIZE / 2; s > 32; s >>= 1)
            {
                if (col < s)
                    partial[col] = max(partial[col], partial[col + s]);

                __syncthreads();
            }

            //if not volatile, must use __synctthreads again, why ?
            volatile float* vpartial = partial;
            if (col < 32)
            {
                vpartial[col] = max(vpartial[col], vpartial[col + 32]);
                vpartial[col] = max(vpartial[col], vpartial[col + 16]);
                vpartial[col] = max(vpartial[col], vpartial[col + 8]);
                vpartial[col] = max(vpartial[col], vpartial[col + 4]);
                vpartial[col] = max(vpartial[col],vpartial[col + 2]);
                vpartial[col] = max(vpartial[col], vpartial[col + 1]);
            }
            
            __syncthreads();

            float max_x = partial[0];

            init = 0;
            for (std::size_t i = col; i < cols; i += step)
                init += exp(x[row * cols + i] - max_x);
        
            partial[col] = init;
            __syncthreads();

            for (std::size_t s = BLOCK_SIZE / 2; s > 32; s >>= 1)
            {
                if (col < s)
                    partial[col] += partial[col + s];

                __syncthreads();
            }

            if (col < 32)
            {
                vpartial[col] += vpartial[col + 32];
                vpartial[col] += vpartial[col + 16];
                vpartial[col] += vpartial[col + 8];
                vpartial[col] += vpartial[col + 4];
                vpartial[col] += vpartial[col + 2];
                vpartial[col] += vpartial[col + 1];
            }


            __syncthreads();

            float logsum = max_x + std::log(partial[0]);
            
            for (std::size_t i = col; i < cols; i += step)
                y[row * cols + i] = x[row * cols + i] - logsum;
        }

    }
    

    void op_log_softmax(const float* x, float* y,
                        std::size_t m, std::size_t n)
    {
        

        auto start = timer::now();

        log_softmax1<<<m, BLOCK_SIZE>>>(x, y, m, n);
        hipDeviceSynchronize();

        auto time = timer::now() - start;
        logs << "[GPU_LOG_SOFTMAX]: " << time << "ms.\n";
    }

}
