#include "hip/hip_runtime.h"
#include "matmul.hh"
#include "../app/mode.hh"
#include "../app/timer.hh"
#include <cmath>
#include <math_functions.h>

namespace gpu
{

    namespace
    {

        constexpr std::size_t BLOCK_SIZE = 512;

        __global__
        void softmax1(const float* x, float* y,
                      std::size_t rows, std::size_t cols) //8ms
        {
            __shared__ float partial[2 * BLOCK_SIZE];

            //load all elements of the array in shared memory
            auto row = blockIdx.x;
            auto col = threadIdx.x;
            std::size_t step = BLOCK_SIZE;

            float init = 1e-30;
            for (std::size_t i = col; i < cols; i += step)
                init = max(x[row * cols + i], init);
        
            partial[col] = init;
            __syncthreads();

            for (std::size_t s = BLOCK_SIZE / 2; s > 32; s >>= 1)
            {
                if (col < s)
                    partial[col] = max(partial[col], partial[col + s]);

                __syncthreads();
            }

            //if not volatile, must use __synctthreads again, why ?
            volatile float* vpartial = partial;
            if (col < 32)
            {
                vpartial[col] = max(vpartial[col], vpartial[col + 32]);
                vpartial[col] = max(vpartial[col], vpartial[col + 16]);
                vpartial[col] = max(vpartial[col], vpartial[col + 8]);
                vpartial[col] = max(vpartial[col], vpartial[col + 4]);
                vpartial[col] = max(vpartial[col],vpartial[col + 2]);
                vpartial[col] = max(vpartial[col], vpartial[col + 1]);
            }
            
            __syncthreads();
            
            for (std::size_t i = col; i < cols; i += step)
                y[row * cols + i] = exp(x[row * cols + i] - partial[0]);

            __syncthreads();

            init = 0;
            for (std::size_t i = col; i < cols; i += step)
                init += y[row * cols + i];
        
            partial[col] = init;
            __syncthreads();

            for (std::size_t s = BLOCK_SIZE / 2; s > 32; s >>= 1)
            {
                if (col < s)
                    partial[col] += partial[col + s];

                __syncthreads();
            }

            if (col < 32)
            {
                vpartial[col] += vpartial[col + 32];
                vpartial[col] += vpartial[col + 16];
                vpartial[col] += vpartial[col + 8];
                vpartial[col] += vpartial[col + 4];
                vpartial[col] += vpartial[col + 2];
                vpartial[col] += vpartial[col + 1];
            }


            __syncthreads();
            
            for (std::size_t i = col; i < cols; i += step)
                y[row * cols + i] /= partial[0];
        }

    }
    

    void op_softmax(const float* x, float* y,
                    std::size_t m, std::size_t n)
    {
        

        auto start = timer::now();

        softmax1<<<m, BLOCK_SIZE>>>(x, y, m, n);
        hipDeviceSynchronize();

        auto time = timer::now() - start;
        logs << "[GPU_SOFTMAX]: " << time << "ms.\n";
    }

}
